#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <string>
#include <cstdio>
#include <vector>
#include <iterator>
#include <iostream>
#include <random>
#include <chrono>
#include <memory>
#include <functional>
#include <Windows.h>

#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<>
#include<conio.h>


using namespace std;

#define N 40 //rozmiar ci�gu binarnergo
#define M 15000 //ilo�� tablic binarnyc h
#define SHOW_DIFFS true //czy pokazywa� r�nic� mi�dzy kolejnymi danymi ci�gami bit�w

#define CUDA_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__device__ unsigned long long counter = 0;

__global__ void cudaHammingDistance2dEquals1(bool *d_arrays, bool *d_pairs)
{
	const long numThreads = blockDim.x * gridDim.x;
	const long threadID = blockIdx.x * blockDim.x + threadIdx.x;
	int i, j;
	bool flag = false;
	for (int ind = threadID; ind < M * M; ind += numThreads)
	{
		i = ind % M;
		j = ind / M;
		if (i == j)
			continue;
		for (int p = 0; p < N; p += 1)
			if (d_arrays[i * N + p] != d_arrays[j * N + p])
				if (flag)
				{
					flag = false;
					break;
				}
				else
					flag = true;
		if (flag)
		{
			d_pairs[i * M + j] = true;
		}
	}
}

__global__ void cudaHammingDistance2d(bool *d_arrays, unsigned long long *d_distances)
{
	const long numThreads = blockDim.x * gridDim.x;
	const long threadID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int ind = threadID; ind < M * M; ind += numThreads)
	{
		int i = ind % M;
		int j = ind / M;
		if (i == j)
			continue;
		for (int p = 0; p < N; p += 1)
			if (d_arrays[i * N + p] != d_arrays[j * N + p])
			{
				d_distances[i * M + j]++;
			}
	}
}

static int m_w = 17358;
static int m_z = 341;

__host__ unsigned int simplerand(void) {
	m_z = 36969 * (m_z & 65535) + (m_z >> 16);
	m_w = 18000 * (m_w & 65535) + (m_w >> 16);
	return (m_z << 16) + m_w;
}



__host__ void ShowGpuResults(bool *distances, bool *bitArrays)
{
	int pairCount = 0;
	HANDLE hConsole = GetStdHandle(STD_OUTPUT_HANDLE);
	cout << "All pairs:\n";
	for (int i = 0; i < M; i++)
		for (int j = 0; j < M; j++)
		{
			if (distances[i * M + j])
			{
				if (pairCount++ > 0)
					cout << ", ";
				cout << "(" << i << ", " << j << ")";
			}
		}
	cout << "\nPair count: " << pairCount << "\n";

	if (SHOW_DIFFS)
	{
		for (int i = 0; i < M; i++)
			for (int j = 0; j < M; j++)
			{
				if (distances[i * M + j])
				{
					cout << "Pair: (" << i << ", " << j << ")\n";

					for (int ind = 0; ind < N; ind++)
					{
						if (bitArrays[i * N + ind] == bitArrays[j * N + ind])
							cout << bitArrays[i * N + ind];
						else
						{
							SetConsoleTextAttribute(hConsole, 12);
							cout << bitArrays[i * N + ind];
							SetConsoleTextAttribute(hConsole, 7);
						}
					}
					cout << "\n";
					for (int ind = 0; ind < N; ind++)
					{
						if (bitArrays[i * N + ind] == bitArrays[j * N + ind])
							cout << bitArrays[j * N + ind];
						else
						{
							SetConsoleTextAttribute(hConsole, 12);
							cout << bitArrays[j * N + ind];
							SetConsoleTextAttribute(hConsole, 7);
						}
					}
					cout << "\n";
				}
			}
	}
}


__host__ void InitializeArrays(bool* arrays, bool randomDistance = false)
{
	std::minstd_rand gen(std::random_device{}());
	std::uniform_real_distribution<double> dist(0, 1);
	if (randomDistance)
		for (long long i = 0; i < M; ++i)
			for (long long j = 0; j < N; ++j)
				arrays[i * N + j] = (((double)simplerand()) / UINT_MAX) > 0.5;
	else
	{
		int* indexes = (int*)calloc(M, sizeof(int));
		int ind = 1;
		while (ind < M)
		{
			int rand = (int)((((double)simplerand()) / UINT_MAX) * N);
			indexes[ind++] = rand;
		}
		for (int i = 0; i < M; i++)
			arrays[i * N + indexes[i]] = true;

		free(indexes);
	}
}

__host__ bool CpuHammingDistance2d(bool *bitArrays)
{
	bool returnFlag = true;
	auto start = chrono::high_resolution_clock::now();
	int* distances = (int*)calloc(M * M, sizeof(int*));
	for (int i = 0; i < M; i++)
		for (int j = 0; j < M; j++)
			for (int p = 0; p < N; p++)
				if (bitArrays[i * N + p] != bitArrays[j * N + p])
					distances[i * M + j]++;
	for (int i = 0; i < M; i++)
		for (int j = 0; j < M; j++)
		{
			if (i == j)
				break;
			if (distances[i * M + j] != 1)
				returnFlag = false;
		}
	free(distances);
	auto finish = chrono::high_resolution_clock::now();
	auto miliseconds = chrono::duration_cast<chrono::milliseconds>(finish - start);
	cout << ">>>>>>CpuHammingDistance2d time: " << miliseconds.count() << " milliseconds\n";
	cout << "Result: " << returnFlag << "\n";
	return returnFlag;
}

__host__ bool GpuHammingDistance2d(bool *bitArrays)
{
	bool returnFlag = true;
	bool *d_bitArrays;
	bool *d_distances, *distances = (bool*)malloc(M * M * sizeof(bool));
	long threadCount = 1024;
	long blockCount = 1024;
	CUDA_CALL(hipSetDevice(0));
	//size_t free = 0, total = 0;
	//hipMemGetInfo(&free, &total);
	CUDA_CALL(hipMalloc((void**)&d_bitArrays, M * N * sizeof(bool)));
	CUDA_CALL(hipMalloc((void**)&d_distances, M * M * sizeof(bool)));
	CUDA_CALL(hipMemset(d_distances, 0, M * M));
	CUDA_CALL(hipMemcpy(d_bitArrays, bitArrays, M * N * sizeof(bool), hipMemcpyHostToDevice));

	auto start = chrono::high_resolution_clock::now();
	CUDA_CALL(hipDeviceSynchronize());
	cudaHammingDistance2dEquals1 << <threadCount, blockCount >> > (d_bitArrays, d_distances);
	// Check for any errors launching the kernel
	CUDA_CALL(hipPeekAtLastError());

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	CUDA_CALL(hipDeviceSynchronize());
	CUDA_CALL(hipMemcpy(distances, d_distances, M * M * sizeof(bool), hipMemcpyDeviceToHost));
	CUDA_CALL(hipFree(d_distances));
	CUDA_CALL(hipFree(d_bitArrays));
	CUDA_CALL(hipDeviceReset());
	ShowGpuResults(distances, bitArrays);
	free(distances);
	auto finish = chrono::high_resolution_clock::now();
	auto milliseconds = chrono::duration_cast<chrono::milliseconds>(finish - start);
	cout << ">>>>>>GpuHammingDistance2d time: " << milliseconds.count() << " milliseconds\n";
	return true;
}

__host__ int main()
{
	auto start = chrono::high_resolution_clock::now();
	//sp�aszczona dwuwymiarowa tablica
	bool* bitArrays = (bool*)calloc(M * N, sizeof(bool));
	auto finish = chrono::high_resolution_clock::now();
	auto miliseconds = chrono::duration_cast<chrono::milliseconds>(finish - start);
	cout << ">>>>>>malloc time: " << miliseconds.count() << " milliseconds\n";

	start = chrono::high_resolution_clock::now();
	InitializeArrays(bitArrays, true);
	finish = chrono::high_resolution_clock::now();
	miliseconds = chrono::duration_cast<chrono::milliseconds>(finish - start);
	cout << ">>>>>>initialize_arrays time: " << miliseconds.count() << " milliseconds\n";
	cout << "Arrays length: " << N << " number of arrays: " << M << "\n";

	CpuHammingDistance2d(bitArrays);
	GpuHammingDistance2d(bitArrays);

	free(bitArrays);
	return 0;
}
